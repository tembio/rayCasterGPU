#include "hip/hip_runtime.h"

#ifndef CUDACODE_H
#define CUDACODE_H

#include "cudaCode.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include <stdio.h>


__global__ void CUDAintersection(int width,int height,float *rays,int numTriangles,float *triangles, float *output){ 

  //CUDA parameters
  int chunkSize = width*height / gridDim.x;
  int rayNumber = chunkSize*blockIdx.x + threadIdx.x;

  float closestT=1000000;
  output[rayNumber*4] = -1;
  output[rayNumber*4 +1] = 0;
  output[rayNumber*4 +2] = 0;
  output[rayNumber*4 +3] = 0;

  for(int p=0;p<numTriangles;p++){

      float  a,b,c,d,e,f,g,h,i,j,k,l,beta,gamma,t;

      //V1 of triangle
      a = triangles[(p*10)+1] - triangles[(p*10)+4];
      b = triangles[(p*10)+2] - triangles[(p*10)+5];
      c = triangles[(p*10)+3] - triangles[(p*10)+6];
      //V2 of triangle
      d = triangles[(p*10)+1] - triangles[(p*10)+7];
      e = triangles[(p*10)+2] - triangles[(p*10)+8];
      f = triangles[(p*10)+3] - triangles[(p*10)+9];
      
      g = rays[(rayNumber*6)+3];
      h = rays[(rayNumber*6)+4];
      i = rays[(rayNumber*6)+5];

      j = triangles[(p*10)+1] - rays[(rayNumber*6)];
      k = triangles[(p*10)+2] - rays[(rayNumber*6)+1];
      l = triangles[(p*10)+3] - rays[(rayNumber*6)+2];

      //Using Kramer to solve system to find barycentric coordinates
      
      beta = (j*(e*i-h*f) + k*(g*f-d*i) + l*(d*h-e*g))/(a*(e*i-h*f)+b*(g*f-d*i)+c*(d*h-e*g));

      gamma = (i*(a*k-j*b)+h*(j*c-a*l)+g*(b*l-k*c))/(a*(e*i-h*f)+b*(g*f-d*i)+c*(d*h-e*g));

      t = -((f*(a*k-j*b)+e*(j*c-a*l)+d*(b*l-k*c))/(a*(e*i-h*f)+b*(g*f-d*i)+c*(d*h-e*g)));

      
      if(t>0){
         //Test if it is closets intersection
         if(t<closestT){
           //test if inside triangle
           if(beta+gamma <=1 && 0<=beta && 0<=gamma){


                     output[rayNumber*4]=2;
             output[(rayNumber*4)+1] =37;
             output[(rayNumber*4)+2] =7;
             output[(rayNumber*4)+3] =57;
             //Calculate intersection point 
             closestT=t;
             output[rayNumber*4]=triangles[(p*10)];
             output[(rayNumber*4)+1] = rays[(rayNumber*6)] + rays[(rayNumber*6)+3]*t;
             output[(rayNumber*4)+2] = rays[(rayNumber*6)+1] + rays[(rayNumber*6)+4]*t;
             output[(rayNumber*4)+3] = rays[(rayNumber*6)+2] + rays[(rayNumber*6)+5]*t;
            }
         }
       }

    }//End calculation per ray

  }

  __global__ void CUDAshading(int width,int height,float *rays,float *interPoints,
                              float *normals,float *materials,
                              float *lights,int numLights,
                              float bgColorR,float bgColorG,float bgColorB,
                              float *pixelColors){
  //---INITIALIZATIONS---

  //CUDA parameters
  int chunkSize = width*height / gridDim.x;
  int pixelNumber = chunkSize*blockIdx.x + threadIdx.x;

  //Direction of the intersecting ray
  float rayDirX = rays[pixelNumber*3],
        rayDirY = rays[pixelNumber*3 + 1],
        rayDirZ = rays[pixelNumber*3 + 2];

  //Find the ID of  the intersected triangle and the intersection point
  int idTriangle = (int)interPoints[pixelNumber*4];

  //If no intersection...assign background color
  if(idTriangle==-1){
      pixelColors[pixelNumber*3] = bgColorR; 
      pixelColors[pixelNumber*3 + 1] = bgColorG;
      pixelColors[pixelNumber*3 + 2] = bgColorB;

  }else{

     float interX = interPoints[pixelNumber*4 + 1],
           interY = interPoints[pixelNumber*4 + 2],
           interZ = interPoints[pixelNumber*4 + 3];
   
     //Material of the intersected triangle
     float colorMatR = materials[idTriangle*5],
           colorMatG = materials[idTriangle*5 + 1],
           colorMatB = materials[idTriangle*5 + 2],
           materialDiffuse = materials[idTriangle*5 + 3],
           materialAmbient = materials[idTriangle*5 + 4];
   
     //Normal of the intersected triangle
     float normalX = normals[idTriangle*3],
           normalY = normals[idTriangle*3 + 1],
           normalZ = normals[idTriangle*3 + 2];
   
     //
     float ambientR = lights[0];
     float ambientG = lights[1];
     float ambientB = lights[2];
   
     //---CALCULATIONS---
   
     //Get ambient component of light
     float lightR=0,lightG=0,lightB=0;
     lightR = ambientR * materialAmbient;
     lightG = ambientG * materialAmbient;
     lightB = ambientB * materialAmbient;
   
     //Normalize normals
     float size=sqrt(normalX*normalX+normalY*normalY+normalZ*normalZ);
     normalX /= size;
     normalY /= size;
     normalZ /= size;
   
     //Calculate normal-light angle
     float costheta = normalX*rayDirX + normalY*rayDirY + normalZ*rayDirZ;
     if(costheta > 0){
         normalX *= -1;
         normalY *= -1;
         normalZ *= -1;
     }
   
     //Lights
     for(int i=0;i<numLights;i++){
   
       float tmpLightR = lights[3 + i*6];
       float tmpLightG = lights[3 + i*6 + 1];
       float tmpLightB = lights[3 + i*6 + 2];
   
       float tmpLightX = lights[3 + i*6 + 3];
       float tmpLightY = lights[3 + i*6 + 4];
       float tmpLightZ = lights[3 + i*6 + 5];
   
       //Calculate intersection-lightsource vector
       float vectorLightX = tmpLightX - interX;
       float vectorLightY = tmpLightY - interY;
       float vectorLightZ = tmpLightZ - interZ;


       float length = sqrt(vectorLightX*vectorLightX+vectorLightY*vectorLightY+vectorLightZ*vectorLightZ);
       vectorLightX /= length;
       vectorLightY /= length;
       vectorLightZ /= length;
   
       float cosphi = normalX*vectorLightX + normalY*vectorLightY + normalZ*vectorLightZ;
   
       if(cosphi>0){
         lightR += tmpLightR*cosphi*materialDiffuse;
         lightG += tmpLightG*cosphi*materialDiffuse;
         lightB += tmpLightB*cosphi*materialDiffuse;
       }
   
     }
   
     pixelColors[pixelNumber*3] = colorMatR * lightR; 
     pixelColors[pixelNumber*3 + 1] = colorMatG * lightB;
     pixelColors[pixelNumber*3 + 2] = colorMatB * lightG;

  }

}



void intersection(int numThreads,int width,int height,float *rays,int numTriangles,float *triangles, float *output){ 

   float *d_rays, *d_triangles, *d_out;

   //Allocate memory in device
   hipMalloc((void **)&d_rays,      6 *height*width*sizeof(float));
   hipMalloc((void **)&d_triangles, 10*numTriangles*sizeof(float));
   hipMalloc((void **)&d_out,       4 *height*width*sizeof(float));

   // copy host matrix to device matrix
   hipMemcpy(d_rays,rays, 6*height*width*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(d_triangles,triangles, 10*numTriangles*sizeof(float), hipMemcpyHostToDevice);

   // check for error
  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }


   int numBlocks=height*width/numThreads;

   dim3 dimBlock(numThreads,1,1); // block size
   dim3 dimGrid(numBlocks,1); // nblocks

   // invoke the kernel
   CUDAintersection<<<dimGrid, dimBlock>>>(width,height,d_rays,numTriangles,d_triangles,d_out);

   hipDeviceSynchronize();

   hipMemcpy(output, d_out, width*height*4*sizeof(float), hipMemcpyDeviceToHost);
 
   hipFree(d_rays);
   hipFree(d_triangles);
   hipFree(d_out);

}



void shading(int numThreads,int width,int height,int numTriangles,int numLights,
             float *rays,float *interPoints,float *normals,
             float *materials,float *lights,
             float bgColorR,float bgColorG,float bgColorB,
             float *pixelColors){

   float *d_rays, *d_inter, *d_normals, *d_materials,*d_lights, *d_pixelColors;

   hipMalloc((void **)&d_rays,       3 *height*width*sizeof(float));
   hipMalloc((void **)&d_inter,4 *height*width*sizeof(float));
   hipMalloc((void **)&d_normals,    3 *numTriangles*sizeof(float));
   hipMalloc((void **)&d_materials,  5 *numTriangles*sizeof(float));
   hipMalloc((void **)&d_lights,    (6 *numLights +3)*sizeof(float));
   hipMalloc((void **)&d_pixelColors,3 *height*width*sizeof(float));
 
   // copy host matrix to device matrix
   hipMemcpy(d_rays, rays, 3*height*width*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(d_inter, interPoints, 4*height*width*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(d_normals, normals, 3*numTriangles*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(d_materials, materials, 5*numTriangles*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(d_lights, lights, (6 *numLights +3)*sizeof(float), hipMemcpyHostToDevice);

   int numBlocks=height*width/numThreads;

   dim3 dimBlock(numThreads,1,1); // block size
   dim3 dimGrid(numBlocks,1); // nblocks

   // invoke the kernel
   CUDAshading<<<dimGrid, dimBlock>>>(width,height,d_rays,d_inter,d_normals,d_materials,d_lights,numLights,bgColorR,bgColorG,bgColorB,d_pixelColors);

   hipDeviceSynchronize();

   hipMemcpy(pixelColors, d_pixelColors, width*height*3*sizeof(float), hipMemcpyDeviceToHost);

       // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }

   hipFree(d_rays);
   hipFree(d_inter);
   hipFree(d_normals);
   hipFree(d_materials);
   hipFree(d_materials);
   hipFree(d_lights);
   hipFree(d_pixelColors);

}























 void intersectionSerie(int width,int height,float *rays,int numTriangles,float *triangles, float *output){ 

  int numRays = width*height;

  for(int it=0;it<numRays;it++){

  int rayNumber=it;

  float rayOriginX= rays[(rayNumber*6)],
        rayOriginY= rays[(rayNumber*6)+1], 
        rayOriginZ= rays[(rayNumber*6)+2],
        rayDirectionX= rays[(rayNumber*6)+3],
        rayDirectionY= rays[(rayNumber*6)+4],
        rayDirectionZ= rays[(rayNumber*6)+5];

  float idIntersect=-1,closestT=1000000,interX=0,interY=0,interZ=0;

  for(int p=0;p<numTriangles;p++){

      float id=  triangles[(p*10)],
            v0x= triangles[(p*10)+1],
            v0y= triangles[(p*10)+2],
            v0z= triangles[(p*10)+3],
            v1x= triangles[(p*10)+4],
            v1y= triangles[(p*10)+5],
            v1z= triangles[(p*10)+6],
            v2x= triangles[(p*10)+7],
            v2y= triangles[(p*10)+8],
            v2z= triangles[(p*10)+9];

      double a,b,c,d,e,f,g,h,i,j,k,l,M,beta,gamma,t;

      //V1 of triangle
      a = v0x - v1x;
      b = v0y - v1y;
      c = v0z - v1z;
      //V2 of triangle
      d = v0x - v2x;
      e = v0y - v2y;
      f = v0z - v2z;
      
      g = rayDirectionX;
      h = rayDirectionY;
      i = rayDirectionZ;

      j = v0x - rayOriginX;
      k = v0y - rayOriginY;
      l = v0z - rayOriginZ;

      //Using Kramer to solve system to find barycentric coordinates
      
      M = a*(e*i-h*f)+b*(g*f-d*i)+c*(d*h-e*g);

      beta = (j*(e*i-h*f) + k*(g*f-d*i) + l*(d*h-e*g))/M;

      gamma = (i*(a*k-j*b)+h*(j*c-a*l)+g*(b*l-k*c))/M;

      t = (f*(a*k-j*b)+e*(j*c-a*l)+d*(b*l-k*c))/M;
      t= -t;

      if(t>0){
        //Test if it is closets intersection
        if(t<closestT){
          //test if inside triangle
          if(beta+gamma <=1 && 0<=beta && 0<=gamma){
            //Calculate intersection point 
            closestT=t;
            idIntersect=id;
            interX = rayOriginX + rayDirectionX*t;
            interY = rayOriginY + rayDirectionY*t;
            interZ = rayOriginZ + rayDirectionZ*t;
           }
        }
      }

    }//End calculation per ray

    output[rayNumber*4] = idIntersect;
    output[(rayNumber*4)+1] = interX;
    output[(rayNumber*4)+2] = interY;
    output[(rayNumber*4)+3] = interZ;

  }

  }




void shadingSerie(int width,int height,float *rays,float *interPoints,
                              float *normals,float *materials,
                              float *lights,int numLights,
                              float bgColorR,float bgColorG,float bgColorB,
                              float *pixelColors){
  //---INITIALIZATIONS---

  int numPixels = width*height;

for(int it=0;it<numPixels;it++){

  int pixelNumber=it;

  //Direction of the intersecting ray
  float rayDirX = rays[pixelNumber*3],
        rayDirY = rays[pixelNumber*3 + 1],
        rayDirZ = rays[pixelNumber*3 + 2];

  //Find the ID of  the intersected triangle and the intersection point
  int idTriangle = (int)interPoints[pixelNumber*4];

  //If no intersection...assign background color
  if(idTriangle==-1){

      pixelColors[pixelNumber*3] = bgColorR; 
      pixelColors[pixelNumber*3 + 1] = bgColorG;
      pixelColors[pixelNumber*3 + 2] = bgColorB;

      continue;
  }

  float interX = interPoints[pixelNumber*4 + 1],
        interY = interPoints[pixelNumber*4 + 2],
        interZ = interPoints[pixelNumber*4 + 3];

  //Material of the intersected triangle
  float colorMatR = materials[idTriangle*5],
        colorMatG = materials[idTriangle*5 + 1],
        colorMatB = materials[idTriangle*5 + 2],
        materialDiffuse = materials[idTriangle*5 + 3],
        materialAmbient = materials[idTriangle*5 + 4];

  //Normal of the intersected triangle
  float normalX = normals[idTriangle*3],
        normalY = normals[idTriangle*3 + 1],
        normalZ = normals[idTriangle*3 + 2];

  //
  float ambientR = lights[0];
  float ambientG = lights[1];
  float ambientB = lights[2];

  //---CALCULATIONS---

  //Get ambient component of light
  float lightR=0,lightG=0,lightB=0;
  lightR = ambientR * materialAmbient;
  lightG = ambientG * materialAmbient;
  lightB = ambientB * materialAmbient;

  //Normalize normals
  float size=sqrt(normalX*normalX+normalY*normalY+normalZ*normalZ);
  normalX /= size;
  normalY /= size;
  normalZ /= size;

  //Calculate normal-light angle
  float costheta = normalX*rayDirX + normalY*rayDirY + normalZ*rayDirZ;
  if(costheta > 0){
      normalX *= -1;
      normalY *= -1;
      normalZ *= -1;
  }

  //Lights
  for(int i=0;i<numLights;i++){

    float tmpLightR = lights[3 + i*6];
    float tmpLightG = lights[3 + i*6 + 1];
    float tmpLightB = lights[3 + i*6 + 2];

    float tmpLightX = lights[3 + i*6 + 3];
    float tmpLightY = lights[3 + i*6 + 4];
    float tmpLightZ = lights[3 + i*6 + 5];

    //Calculate intersection-lightsource vector
    float vectorLightX = tmpLightX - interX;
    float vectorLightY = tmpLightY - interY;
    float vectorLightZ = tmpLightZ - interZ;

    float length = sqrt(vectorLightX*vectorLightX+vectorLightY*vectorLightY+vectorLightZ*vectorLightZ);
    vectorLightX /= length;
    vectorLightY /= length;
    vectorLightZ /= length;

    float cosphi = normalX*vectorLightX + normalY*vectorLightY + normalZ*vectorLightZ;

    if(cosphi>0){
      lightR += tmpLightR*cosphi*materialDiffuse;
      lightG += tmpLightG*cosphi*materialDiffuse;
      lightB += tmpLightB*cosphi*materialDiffuse;
    }

  }

  pixelColors[pixelNumber*3] = colorMatR * lightR; 
  pixelColors[pixelNumber*3 + 1] = colorMatG * lightB;
  pixelColors[pixelNumber*3 + 2] = colorMatB * lightG;
}

}

















#endif